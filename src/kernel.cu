
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define min(x, y) (x > y ? x : y)
#define N 33 * 1024

#define ThreadPerBlock 256

// smallest multiple of threadsPerBlock that is greater than or equal to N
#define blockPerGrid min(32, (N + ThreadPerBlock - 1) / ThreadPerBlock)

__global__ void Vector_Dot_Product(const float *V1, const float *V2,
                                   float *V3) {
  __shared__ float chache[ThreadPerBlock];

  float temp;

  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  const unsigned int chacheindex = threadIdx.x;

  while (tid < N) {
    temp += V1[tid] * V2[tid];

    tid += blockDim.x * gridDim.x;
  }

  chache[chacheindex] = temp;

  __syncthreads();

  int i = blockDim.x / 2;

  while (i != 0) {
    if (chacheindex < i) chache[chacheindex] += chache[chacheindex + i];

    __syncthreads();

    i /= 2;
  }

  if (chacheindex == 0) V3[blockIdx.x] = chache[0];
}

void real_main(void) {
  float *V1_H, *V2_H, *V3_H;
  float *V1_D, *V2_D, *V3_D;

  V1_H = new float[N];
  V2_H = new float[N];
  V3_H = new float[blockPerGrid];

  hipMalloc((void **)&V1_D, N * sizeof(float));

  hipMalloc((void **)&V2_D, N * sizeof(float));

  hipMalloc((void **)&V3_D, blockPerGrid * sizeof(float));

  for (int i = 0; i < N; i++) {
    V1_H[i] = i;

    V2_H[i] = i * 2;
  }

  hipMemcpy(V1_D, V1_H, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(V2_D, V2_H, N * sizeof(float), hipMemcpyHostToDevice);

  Vector_Dot_Product<<<blockPerGrid, ThreadPerBlock>>>(V1_D, V2_D, V3_D);

  hipMemcpy(V3_H, V3_D, N * sizeof(float), hipMemcpyDeviceToHost);

  cout << "\n Vector Dot Prodcut is : ";

  float sum = 0;

  for (int i = 0; i < blockPerGrid; i++) sum += V3_H[i];
  cout << sum << endl;

  hipFree(V1_D);
  hipFree(V2_D);
  hipFree(V3_D);

  delete[] V1_H;
  delete[] V2_H;
  delete[] V3_H;
}